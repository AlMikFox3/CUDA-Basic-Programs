#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>

#include <stdio.h>

__global__ void addKernel(int * dev_a, int * dev_b, int * dev_c)
{
	int i = threadIdx.x;
	dev_c[i] = dev_a[i] + dev_b[i];
}

int main()
{
	const int size = 3;
	int a[size][size];
	int b[size][size];
	int c[size][size] = { 0 };
	int i = 0, j = 0;
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
			scanf("%d", &a[i][j]);
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
			scanf("%d", &b[i][j]);

	int *dev_a, *dev_b, *dev_c;
	int t = size*size*sizeof(int);
	hipMalloc((void**)&dev_a, t);
	hipMalloc((void**)&dev_b, t);
	hipMalloc((void**)&dev_c, t);
	hipMemcpy(dev_a, a, t, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, t, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, t, hipMemcpyHostToDevice);
	addKernel << <1, size*size >> >(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, t, hipMemcpyDeviceToHost);
	printf("-----OUTPUT-----\n");
	for (i = 0; i < size; i++){
		for (j = 0; j < size; j++){
			printf("%d ", c[i][j]);
		}
		printf("\n");
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}



