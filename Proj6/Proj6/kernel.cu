#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>

#include <stdio.h>

__global__ void addKernel(int * dev_a, int* x)
{
	int i = threadIdx.x;
	if (dev_a[i] < *x)
		dev_a[i] = 0;
	else
		dev_a[i] = 1;
}

int main()
{
	const int size = 6;
	int a[size][size];
	int b[size][size] = { 0 };
	int i = 0, j = 0, x;
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
			scanf("%d", &a[i][j]);

	scanf("%d", &x);

	int *dev_a, *dev_x;
	int t = size*size*sizeof(int);
	hipMalloc((void**)&dev_a, t);
	hipMalloc((void**)&dev_x, sizeof(int));
	hipMemcpy(dev_a, a, t, hipMemcpyHostToDevice);
	hipMemcpy(dev_x, &x, sizeof(int), hipMemcpyHostToDevice);
	addKernel << <1, size*size >> >(dev_a, dev_x);
	hipMemcpy(b, dev_a, t, hipMemcpyDeviceToHost);
	printf("-----OUTPUT-----\n");
	for (i = 0; i < size; i++){
		for (j = 0; j < size; j++){
			printf("%d ", b[i][j]);
		}
		printf("\n");
	}

	return 0;
}



