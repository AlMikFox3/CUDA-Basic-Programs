#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>

#include <stdio.h>

__global__ void addKernel(int * dev_a, int* dev_b ,int* dev_size)
{
	int i = threadIdx.x;
	int j,p;
	for (j = 0; j < (*dev_size); j++)
	{
		p = *dev_size*i + j;
		dev_b[i] += dev_a[p];
		//printf("%d %d\n", i, p);
	}
}

int main()
{
	const int size = 3;
	int s = size;
	int a[size][size];
	int b[size] = { 0 };
	int i = 0, j = 0;
	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++)
			scanf("%d", &a[i][j]);

	int *dev_a, *dev_b, *dev_size;
	int t = size*size*sizeof(int);
	int t1 = size*sizeof(int);
	hipMalloc((void**)&dev_a, t);
	hipMalloc((void**)&dev_b, t);
	hipMalloc((void**)&dev_size, sizeof(int));
	hipMemcpy(dev_a, a, t, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, t1, hipMemcpyHostToDevice);
	hipMemcpy(dev_size, &s, sizeof(int), hipMemcpyHostToDevice);
	addKernel << <1, size >> >(dev_a, dev_b, dev_size);
	hipMemcpy(b, dev_b, t1, hipMemcpyDeviceToHost);
	printf("-----OUTPUT-----\n");
	int p = 0;
	for (i = 0; i < size; i++){
			//printf("%d ", b[i]);
			p += b[i];
		//printf("\n");
	}
	printf("%d", p);
	hipFree(dev_a);
	hipFree(dev_b);
	return 0;
}



